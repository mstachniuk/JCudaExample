
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while(i < n) {
		sum[i] = a[i] + b[i];
	    i = i + blockDim.x * gridDim.x;
    }
}
